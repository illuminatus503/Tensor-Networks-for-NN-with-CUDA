#include <stdlib.h>
#include <stdint.h>
#include <stdbool.h>
#include <assert.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../include/cuda_errchk.cuh"

#include "../include/tq_mem_datatype.h"
#include "../include/tq_mem_gpu.cuh"

#define is_power_of_two(x) ((x & (x - 1)) == 0)

#ifndef DEFAULT_ALIGNMENT
#define DEFAULT_ALIGNMENT (2 * sizeof(void *))
#endif

uintptr_t CUDA_align_forward(uintptr_t ptr, size_t align)
{
    uintptr_t p, a, modulo;

    assert(is_power_of_two(align));

    p = ptr;
    a = (uintptr_t)align;

    // Same as (p % a) but faster as 'a' is a power of two
    modulo = p & (a - 1);

    if (modulo != 0)
    {
        // If 'p' address is not aligned, push the address to the
        // next value which is aligned
        p += a - modulo;
    }

    return p;
}

void *CUDA_arena_alloc_align(Arena *a,
                             size_t size,
                             size_t align)
{
    // Return NULL if the arena is out of memory (or handle differently)
    void *ptr = NULL;

    // Align 'curr_offset' forward to the specified alignment
    uintptr_t curr_ptr = (uintptr_t)a->buf + (uintptr_t)a->curr_offset;
    uintptr_t offset = CUDA_align_forward(curr_ptr, align);
    offset -= (uintptr_t)a->buf; // Change to relative offset

    // Check to see if the backing memory has space left
    if (offset + size <= a->buf_len)
    {
        ptr = &(a->buf[offset]);
        a->prev_offset = offset;
        a->curr_offset = offset + size;

        // Zero new memory by default
        gpuErrchk(
            hipMemset((void *)ptr, 0, size));
    }

    return ptr;
}

// Because C doesn't have default parameters
void *CUDA_arena_alloc(Arena *a, size_t size)
{
    return CUDA_arena_alloc_align(a, size, DEFAULT_ALIGNMENT);
}

void CUDA_arena_init(Arena *a,
                     void *backing_buffer,
                     size_t backing_buffer_length)
{
    a->buf = (unsigned char *)backing_buffer;
    a->buf_len = backing_buffer_length;
    a->curr_offset = 0;
    a->prev_offset = 0;
}

void *CUDA_arena_resize_align(Arena *a,
                              void *old_memory,
                              size_t old_size,
                              size_t new_size,
                              size_t align)
{
    unsigned char *old_mem = (unsigned char *)old_memory;

    assert(is_power_of_two(align));

    if (old_mem == NULL || old_size == 0)
    {
        return CUDA_arena_alloc_align(a, new_size, align);
    }
    else if (a->buf <= old_mem && old_mem < a->buf + a->buf_len)
    {
        if (a->buf + a->prev_offset == old_mem)
        {
            a->curr_offset = a->prev_offset + new_size;
            if (new_size > old_size)
            {
                // Zero the new memory by default
                gpuErrchk(
                    hipMemset(&a->buf[a->curr_offset], 0, new_size - old_size));
            }
            return old_memory;
        }
        else
        {
            void *new_memory = CUDA_arena_alloc_align(a, new_size, align);
            size_t copy_size = old_size < new_size ? old_size : new_size;

            // Copy across old memory to the new memory
            gpuErrchk(
                hipMemcpy((void *)new_memory, (const void *)old_memory,
                           copy_size,
                           hipMemcpyDeviceToDevice));

            return new_memory;
        }
    }
    else
    {
        assert(0 && "Memory is out of bounds of the buffer in this arena");
        return NULL;
    }
}

// Because C doesn't have default parameters
void *CUDA_arena_resize(Arena *a,
                        void *old_memory, size_t old_size,
                        size_t new_size)
{
    return CUDA_arena_resize_align(a, old_memory, old_size, new_size, DEFAULT_ALIGNMENT);
}

void CUDA_arena_free_all(Arena *a)
{
    a->curr_offset = 0;
    a->prev_offset = 0;
}
